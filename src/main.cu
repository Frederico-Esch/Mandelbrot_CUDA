#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <Window.h>

namespace image {
    extern "C" {
        #include <stbi_wrapper.h>
    }

}

struct Pixel { unsigned char r; unsigned char g; unsigned char b; unsigned char a; };
struct State {
    struct { float x; float y; } scale;
    struct { float x; float y; } trans;
    float iterations_mag;
    float divergency_sq;
    float speed;
};
struct hipComplex {
    float real; float imag;
    __device__ void square() {
        auto n_real = (real*real - imag*imag);
        auto n_imag = 2*(real*imag);

        real = n_real;
        imag = n_imag;
    }
    __device__ hipComplex operator+(hipComplex&& c) {
        return hipComplex { real + c.real, imag + c.imag };
    }
    __device__ hipComplex operator+(const hipComplex& c) {
        return hipComplex { real + c.real, imag + c.imag };
    }
    __device__ float mag_sq() { return real*real + imag*imag; }
};
struct Complex {
    float real; float imag;
    void square() {
        auto n_real = (real*real - imag*imag);
        auto n_imag = 2*(real*imag);

        real = n_real;
        imag = n_imag;
    }
    Complex operator+(hipComplex&& c) {
        return Complex { real + c.real, imag + c.imag };
    }
    Complex operator+(const Complex& c) {
        return Complex { real + c.real, imag + c.imag };
    }
    float mag_sq() { return real*real + imag*imag; }
};


#define WIDTH (1900)
#define HEIGHT (900)

//#define TRANSX (WIDTH/1.7f)
//#define TRANSY (HEIGHT*3.0f/1.f)
//#define TRANSX (0.52f) 52e-2
//#define TRANSY (0.51f) 51e-2

//#define SCALEX (0.075f)
//#define SCALEY (0.15f)
//#define SCALEX (0.00015f) 15e-5 ou 15e-6
//#define SCALEY (0.00015f) 15e-5 ou 15e-6

#define RED     (Pixel { 255,   0,   0, 255 })
#define GREEN   (Pixel {   0, 255,   0, 255 })
#define BLUE    (Pixel {   0,   0, 255, 255 })
#define YELLOW  (Pixel { 255, 255,   0, 255 })
#define BLACK   (Pixel {   0,   0,   0, 255 })
#define WHITE   (Pixel { 255, 255, 255, 255 })

//#define ITERATIONS_MAG (500)
//#define DIVERGENCY_SQ (1e30)

__device__
int cuMandelbrot(size_t _x, size_t _y, State state) {

    auto trans = state.trans;
    auto scale = state.scale;
    auto iterations_mag = state.iterations_mag;
    auto divergency_sq = state.divergency_sq;

    //float x = SCALEX * ((float)_x - TRANSX) / (WIDTH/2.f);
    //float y = SCALEY * ((float)_y - TRANSY) / (HEIGHT/2.f);
    float x = scale.x * ((float)_x - (WIDTH/2.f)) / (WIDTH/2.f)  - trans.x;
    float y = scale.y * ((float)_y - (HEIGHT/2.f)) / (HEIGHT/2.f) - trans.y;

    hipComplex Z { 0, 0 };
    for (int i = 0; i < 3*iterations_mag; i ++) {
        Z.square();
        Z = Z + hipComplex { x, y };
        if (Z.mag_sq() > divergency_sq) {
            return i;
        }
    }
    return 0;
}

int mandelbrot(size_t _x, size_t _y, State state) {

    auto scale = state.scale;
    auto trans = state.trans;
    auto iterations_mag = state.iterations_mag;
    auto divergency_sq = state.divergency_sq;

    //float x = SCALEX * ((float)_x - TRANSX) / (WIDTH/2.f);
    //float y = SCALEY * ((float)_y - TRANSY) / (HEIGHT/2.f);
    float x = scale.x * ((float)_x - (WIDTH /2.f)) / (WIDTH/2.f)  - trans.x;
    float y = scale.y * ((float)_y - (HEIGHT/2.f)) / (HEIGHT/2.f) - trans.y;

    Complex Z { 0, 0 };
    for (int i = 0; i < 3*iterations_mag; i ++) {
        Z.square();
        Z = Z + Complex { x, y };
        if (Z.mag_sq() > divergency_sq) {
            return i;
        }
    }
    return 0;
}


__global__
void gpu_draw(Pixel* data, size_t N, State state) {

    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    float iterations_mag = state.iterations_mag;

    while (idx < N) {
        auto divergence = cuMandelbrot( idx % WIDTH, idx / WIDTH, state);

        if (divergence) {
            if (divergence < iterations_mag) {
                data[idx] = BLACK;
                data[idx].r = (unsigned char) (255 * ((float)divergence/iterations_mag));
            }
            else if (divergence < 2*iterations_mag) {
                data[idx] = RED;
                data[idx].g = (unsigned char) (255 * ((float)divergence/iterations_mag - 1));
            }
            else if (divergence < 3*iterations_mag) {
                data[idx] = YELLOW;
                data[idx].b = (unsigned char) (255 * ((float)divergence/iterations_mag - 2));
            }
        }
        else {
            data[idx] = BLUE;
        }

        idx += blockDim.x * gridDim.x;
    }
}

void cpu_draw(Pixel* data, size_t N, State state) {
    float iterations_mag = state.iterations_mag;

    for (size_t i = 0; i < WIDTH; i++) {
        for (size_t j = 0; j < HEIGHT; j++) {
            size_t idx = i + j*WIDTH;
            auto divergence = mandelbrot(i, j, state);

            if (divergence) {

                if (divergence < iterations_mag) {
                    data[idx] = BLACK;
                    data[idx].r = (unsigned char) (255 * ((float)divergence/iterations_mag));
                }
                else if (divergence < 2*iterations_mag) {
                    data[idx] = RED;
                    data[idx].g = (unsigned char) (255 * ((float)divergence/iterations_mag - 1));
                }
                else if (divergence < 3*iterations_mag) {
                    data[idx] = YELLOW;
                    data[idx].b = (unsigned char) (255 * ((float)divergence/iterations_mag - 2));
                }
            }
            else {
                data[idx] = BLUE;
            }
        }
    }
}

int main(void) {

    std::string filename = "teste.png";

    size_t N = WIDTH*HEIGHT;
    auto data = new Pixel[N];
#define GPU

    State state {
        { 15e-5, 15e-5 },
        { 52e-2, 51e-2 },
        500,
        1e30,
    };
    state.speed = std::pow(10, std::log10(state.scale.x) - 1);

#ifdef GPU
    Pixel* dev_data;
    hipMalloc(&dev_data, sizeof(Pixel)*N);
    clock_t start, end;

    dim3 grids(1000);
    dim3 threads(250);

    gpu_draw<<<grids, threads>>>(dev_data, N, state);
    hipMemcpy(data, dev_data, sizeof(Pixel)*N, hipMemcpyDeviceToHost);

#else
    cpu_draw(data, N, state);
#endif

    Window::Init(WIDTH, HEIGHT, "teste");

    Window::LoadImage(WIDTH, HEIGHT, data);

    namespace Gui = Window::Gui;

    while (!Window::ShouldClose()) {

        Window::InitUpdate();
        bool reload_image = false;

        Window::Key pressed = Window::KeyboardPressed();
        if (pressed != Window::Key::NONE) {
            if ((int)pressed & (int)Window::Key::W_KEY) {
                state.trans.y -= state.speed;
            }
            if ((int)pressed & (int)Window::Key::A_KEY) {
                state.trans.x += state.speed;
            }
            if ((int)pressed & (int)Window::Key::S_KEY) {
                state.trans.y += state.speed;
            }
            if ((int)pressed & (int)Window::Key::D_KEY) {
                state.trans.x -= state.speed;
            }

            if ((int)pressed & (int)Window::Key::PLUS_KEY) {
                state.scale.x -= state.speed;
                state.scale.y -= state.speed;

                state.speed = std::pow(10, std::log10(state.scale.x)-1);
            }
            if ((int)pressed & (int)Window::Key::MINUS_KEY) {
                state.scale.x += state.speed;
                state.scale.y += state.speed;

                state.speed = std::pow(10, std::log10(state.scale.x)-1);
            }
            reload_image = true;
        }

        Window::Draw();

        Gui::NewFrame();
        Gui::Begin("Teste");
        Gui::Float("Divergency", &state.divergency_sq);
        Gui::Float("Iterations", &state.iterations_mag);
        Gui::DisplayFloat2("Translation", (float*)&state.trans);
        Gui::DisplayFloat2("Scale", (float*)&state.scale);
        if (Gui::Button("Reload Image")) reload_image = true;
        Gui::End();
        Gui::Render();

        Window::EndUpdate();

        if (reload_image) {
            gpu_draw<<<grids, threads>>>(dev_data, N, state);
            hipMemcpy(data, dev_data, sizeof(Pixel)*N, hipMemcpyDeviceToHost);
            Window::LoadImage(WIDTH, HEIGHT, data);
            reload_image = false;
        }
    }


    //setup dear Im gui thing so that I can move around change the zoom (scale) and variables like iteration_mag and diversion_sq

    //printf("%d\n", image::RGBA(filename.c_str(), WIDTH, HEIGHT, data));
    delete[] data;

    return 0;
}
